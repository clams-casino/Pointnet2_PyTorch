#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// input: new_xyz(b, m, 3) xyz(b, n, 3)
// output: idx(b, m, nsample)
template <typename scalar_t>
__global__ void query_ball_point_kernel(int b, int n, int m, float radius,
                                        int nsample,
                                        const scalar_t *__restrict__ new_xyz,
                                        const scalar_t *__restrict__ xyz,
                                        int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    scalar_t new_x = new_xyz[j * 3 + 0];
    scalar_t new_y = new_xyz[j * 3 + 1];
    scalar_t new_z = new_xyz[j * 3 + 2];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      scalar_t x = xyz[k * 3 + 0];
      scalar_t y = xyz[k * 3 + 1];
      scalar_t z = xyz[k * 3 + 2];
      scalar_t d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) +
                 (new_z - z) * (new_z - z);
      if (d2 < radius2) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

void query_ball_point_kernel_wrapper(int b, int n, int m, float radius,
                                     int nsample, at::Tensor new_xyz,
                                     at::Tensor xyz, int *idx); {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES(xyz.type(), "query_ball_point_kernel", ([&] {
    query_ball_point_kernel<scalar_t><<<b, opt_n_threads(m), 0, stream>>>(
        b, n, m, radius, nsample,
        new_xyz.data_ptr<scalar_t>(),
        xyz.data_ptr<scalar_t>(),
        idx);
  }));


  CUDA_CHECK_ERRORS();
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// input: points(b, c, n) idx(b, npoints, nsample)
// output: out(b, c, npoints, nsample)
template <typename scalar_t>
__global__ void group_points_kernel(int b, int c, int n, int npoints,
                                    int nsample,
                                    const scalar_t *__restrict__ points,
                                    const int *__restrict__ idx,
                                    scalar_t *__restrict__ out) {
  int batch_index = blockIdx.x;
  points += batch_index * n * c;
  idx += batch_index * npoints * nsample;
  out += batch_index * npoints * nsample * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * npoints; i += stride) {
    const int l = i / npoints;
    const int j = i % npoints;
    for (int k = 0; k < nsample; ++k) {
      int ii = idx[j * nsample + k];
      out[(l * npoints + j) * nsample + k] = points[l * n + ii];
    }
  }
}

void group_points_kernel_wrapper(int b, int c, int n, int npoints, int nsample,
                                 at::Tensor points, const int *idx,
                                 at::Tensor &out) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // group_points_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>( #TODO remove this
  //     b, c, n, npoints, nsample, points, idx, out);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(points.type(), "group_points", ([&] {
    group_points_kernel<scalar_t><<<b, opt_block_config(npoints, c), 0, stream>>>(
        b, c, n, npoints, nsample,
        points.data_ptr<scalar_t>(),
        idx,
        out.data_ptr<scalar_t>());
  }));

  CUDA_CHECK_ERRORS();
}

// input: grad_out(b, c, npoints, nsample), idx(b, npoints, nsample)
// output: grad_points(b, c, n)
template <typename scalar_t>
__global__ void group_points_grad_kernel(int b, int c, int n, int npoints,
                                         int nsample,
                                         const scalar_t *__restrict__ grad_out,
                                         const int *__restrict__ idx,
                                         scalar_t *__restrict__ grad_points) {
  int batch_index = blockIdx.x;
  grad_out += batch_index * npoints * nsample * c;
  idx += batch_index * npoints * nsample;
  grad_points += batch_index * n * c;

  const int index = threadIdx.y * blockDim.x + threadIdx.x;
  const int stride = blockDim.y * blockDim.x;
  for (int i = index; i < c * npoints; i += stride) {
    const int l = i / npoints;
    const int j = i % npoints;
    for (int k = 0; k < nsample; ++k) {
      int ii = idx[j * nsample + k];
      atomicAdd(grad_points + l * n + ii,
                grad_out[(l * npoints + j) * nsample + k]);
    }
  }
}

void group_points_grad_kernel_wrapper(int b, int c, int n, int npoints,
                                      int nsample, at::Tensor grad_out,
                                      const int *idx, at::Tensor &grad_points) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // group_points_grad_kernel<<<b, opt_block_config(npoints, c), 0, stream>>>(
  //     b, c, n, npoints, nsample, grad_out, idx, grad_points);


  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_out.type(), "group_points_grad", ([&] {
    group_points_grad_kernel<scalar_t><<<b, opt_block_config(npoints, c), 0, stream>>>(
        b, c, n, npoints, nsample,
        grad_out.data_ptr<scalar_t>(),
        idx,
        grad_points.data_ptr<scalar_t>());
  }));

  CUDA_CHECK_ERRORS();
}
